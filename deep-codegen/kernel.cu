#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// SpMMv Kernel
__global__ void spmmv_kernel(float* input, float* output, int* rowPtr, int* colInd, float* values, int* degrees, int F_in, int V) {
    // Output: node v - feature f
    int v = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < V && f < F_in) {
        int row_start = rowPtr[v];
        int row_end = rowPtr[v + 1];
        float result = 0.0f;
        for (int i = row_start; i < row_end; ++i) {
            int col = colInd[i];
            float val = values[i];
            result += val * input[col * F_in + f];
        }
        output[v * F_in + f] = result / degrees[v];
    }
}

// SpMMv Function
void spmmv(array2d_t<float>& input, array2d_t<float>& output, array1d_t<int>& rowPtr, array1d_t<int>& colInd, array1d_t<float>& values, array1d_t<int>& degrees, int V, int F_in, int64_t stream_id, bool print_stream){
    // Dense input and output
    float* ds_in = input.data_ptr;
    float* out = output.data_ptr;
    // Sparse input in csr
    int* row = rowPtr.data_ptr;
    int* col = colInd.data_ptr;
    float* val = values.data_ptr;
    int* d = degrees.data_ptr;
    // Kernel config
    dim3 blockSize(32, 4);
    dim3 gridSize((F_in + blockSize.x -1) / blockSize.x, (V + blockSize.y - 1) / blockSize.y);
    // Convert int64_t to hipStream_t
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_id);
    // Print the stream ID
    if (print_stream) printf("CUDA Stream ID: %p\n", reinterpret_cast<void*>(stream));
    // Load kernel on the stream
    spmmv_kernel<<<gridSize, blockSize, 0, stream>>>(ds_in, out, row, col, val, d, F_in, V);
}
